#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
 
#include <stdio.h>
#include "cuosd_kernel.h"

typedef unsigned char uint8_t;

#define CUOSD_PRINT_E(f_, ...) \
  fprintf(stderr, "[cuOSD Error] at %s:%d : " f_, (const char*)__FILE__, __LINE__, ##__VA_ARGS__)

#define CUOSD_PRINT_W(f_, ...) \
  printf("[cuOSD Warning] at %s:%d : " f_, (const char*)__FILE__, __LINE__, ##__VA_ARGS__)

template<typename _T> _T max(_T a, _T b){return a >= b ? a : b;}
template<typename _T> _T min(_T a, _T b){return a <= b ? a : b;}

template<typename _T>
static __host__ __device__ unsigned char u8cast(_T value) {
    return value < 0 ? 0 : (value > 255 ? 255 : value);
}

static __host__ __device__ unsigned int round_down2(unsigned int num) {
    return num & (~1);
}

template<typename _T>
static __forceinline__ __device__ _T limit(_T value, _T low, _T high){
    return value < low ? low : (value > high ? high : value);
}

#define INTER_RESIZE_COEF_BITS 11
#define INTER_RESIZE_COEF_SCALE (1 << INTER_RESIZE_COEF_BITS)

static __device__ void __forceinline__ yuv2rgb(
    int y, int u, int v, uint8_t& r, uint8_t& g, uint8_t& b
){
    int iyval = 1220542*max(0, y - 16);
    r = u8cast((iyval + 1673527*(v - 128)                      + (1 << 19)) >> 20);
    g = u8cast((iyval - 852492*(v - 128) - 409993*(u - 128)    + (1 << 19)) >> 20);
    b = u8cast((iyval                      + 2116026*(u - 128) + (1 << 19)) >> 20);
}

static __device__ void __forceinline__ rgb2yuv(
    int r, int g, int b, uint8_t& y, uint8_t& u, uint8_t& v
){
    y = ((66 * r + 129 * g + 25 * b + 128) >> 8) + 16;
    u = ((-38 * r - 74 * g + 112 * b + 128) >> 8) + 128;
    v = ((112 * r - 94 * g - 18 * b + 128) >> 8) + 128;
    // y = u8cast(0.299f * r + 0.587f * g + 0.114f * b);
    // u = u8cast(-0.1687f * r - 0.3313 * g + 0.5f * b + 128);
    // v = u8cast(0.5f * r - 0.4187f * g - 0.0813f * b + 128);
}

// inbox_single_pixel:
// check if given coordinate is in box
//      a --- d
//      |     |
//      b --- c
static __device__ __forceinline__ bool inbox_single_pixel(
    float ix, float iy, float ax, float ay, float bx, float by, float cx, float cy, float dx, float dy) {
    return  ((bx-ax) * (iy - ay) - (by-ay) * (ix-ax)) < 0 &&
            ((cx-bx) * (iy - by) - (cy-by) * (ix-bx)) < 0 &&
            ((dx-cx) * (iy - cy) - (dy-cy) * (ix-cx)) < 0 &&
            ((ax-dx) * (iy - dy) - (ay-dy) * (ix-dx)) < 0;
}

static __device__ void blend_single_color(uchar4& color, unsigned char& c0, unsigned char& c1, unsigned char& c2, unsigned char a) {
    int foreground_alpha = a;
    int background_alpha = color.w;
    int blend_alpha      = ((background_alpha * (255 - foreground_alpha))>> 8) + foreground_alpha;
    color.x = u8cast((((color.x * background_alpha * (255 - foreground_alpha))>>8) + (c0 * foreground_alpha)) / blend_alpha);
    color.y = u8cast((((color.y * background_alpha * (255 - foreground_alpha))>>8) + (c1 * foreground_alpha)) / blend_alpha);
    color.z = u8cast((((color.z * background_alpha * (255 - foreground_alpha))>>8) + (c2 * foreground_alpha)) / blend_alpha);
    color.w = blend_alpha;
}

CircleCommand::CircleCommand(int cx, int cy, int radius, int thickness, unsigned char c0, unsigned char c1, unsigned char c2, unsigned char c3) {
    this->type = CommandType::Circle;
    this->cx = cx;
    this->cy = cy;
    this->radius    = radius;
    this->thickness = thickness;
    this->c0 = c0;
    this->c1 = c1;
    this->c2 = c2;
    this->c3 = c3;

    int half_thickness = (thickness + 1) / 2 + 2;
    this->bounding_left  = cx - radius - half_thickness;
    this->bounding_right = cx + radius + half_thickness;
    this->bounding_top   = cy - radius - half_thickness;
    this->bounding_bottom = cy + radius + half_thickness;
}

RectangleCommand::RectangleCommand() {
    this->type = CommandType::Rectangle;
}

BoxBlurCommand::BoxBlurCommand(){
    this->type = CommandType::BoxBlur;
}

TextCommand::TextCommand(int text_line_size, int ilocation, unsigned char c0, unsigned char c1, unsigned char c2, unsigned char c3) {
    this->text_line_size = text_line_size;
    this->ilocation      = ilocation;
    this->type           = CommandType::Text;
    this->c0 = c0;
    this->c1 = c1;
    this->c2 = c2;
    this->c3 = c3;
}

SegmentCommand::SegmentCommand() {
    this->type = CommandType::Segment;
}

PolyFillCommand::PolyFillCommand() {
    this->type = CommandType::PolyFill;
}

RGBASourceCommand::RGBASourceCommand() {
    this->type = CommandType::RGBASource;
}

NV12SourceCommand::NV12SourceCommand() {
    this->type = CommandType::NV12Source;
}

// interpolation_fn:
// interpolate alpha for border pixels
static __device__ unsigned char interpolation_fn(
    float x, int a, int b, int padding, unsigned char origin_alpha
) {
    int x0 = a - padding < 0 ? 0 : a - padding;
    int x1 = b + padding;
    if (x < x0 || x > x1) return 0;
    if (x >= a && x < b) return origin_alpha;
    if (x >= b && x <= x1) return (x1 - x) / padding * origin_alpha;
    if (x < a && x >= x0) return (x - x0) / padding * origin_alpha;
    return 0;
}

// external_msaa4x:
// check if given coordinate is on border or outside the border, do msaa4x for border pixels
static __device__ __forceinline__ bool external_msaa4x(
    float ix, float iy, float ax, float ay, float bx, float by, float cx, float cy, float dx, float dy,
    unsigned char a, unsigned char& alpha) {
    bool h0 = !inbox_single_pixel(ix-0.25f, iy-0.25f, ax, ay, bx, by, cx, cy, dx, dy);
    bool h1 = !inbox_single_pixel(ix+0.25f, iy-0.25f, ax, ay, bx, by, cx, cy, dx, dy);
    bool h2 = !inbox_single_pixel(ix+0.25f, iy+0.25f, ax, ay, bx, by, cx, cy, dx, dy);
    bool h3 = !inbox_single_pixel(ix-0.25f, iy+0.25f, ax, ay, bx, by, cx, cy, dx, dy);
    if (h0 || h1 || h2 || h3) {
        if (h0 && h1 && h2 && h3) return true;
        alpha = a * (h0 + h1 + h2 + h3) * 0.25f;
        return true;
    }
    return false;
}

// internal_msaa4x:
// check if given coordinate is on border or inside the border, do msaa4x for border pixels
static __device__ __forceinline__ bool internal_msaa4x(
    float ix, float iy, float ax, float ay, float bx, float by, float cx, float cy, float dx, float dy,
    unsigned char a, unsigned char& alpha) {
    bool h0 = inbox_single_pixel(ix-0.25f, iy-0.25f, ax, ay, bx, by, cx, cy, dx, dy);
    bool h1 = inbox_single_pixel(ix+0.25f, iy-0.25f, ax, ay, bx, by, cx, cy, dx, dy);
    bool h2 = inbox_single_pixel(ix+0.25f, iy+0.25f, ax, ay, bx, by, cx, cy, dx, dy);
    bool h3 = inbox_single_pixel(ix-0.25f, iy+0.25f, ax, ay, bx, by, cx, cy, dx, dy);
    if (h0 || h1 || h2 || h3) {
        alpha = a * (h0 + h1 + h2 + h3) * 0.25f;
        return true;
    }
    return false;
}

// render_rectangle_fill_msaa4x:
// render filled rectangle with border msaa4x interpolation on
static __device__ void render_rectangle_fill_msaa4x(int ix, int iy, RectangleCommand* p, uchar4 color[4]) {
    unsigned char alpha;
    if (internal_msaa4x(ix, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1, p->c3, alpha)) {
        blend_single_color(color[0], p->c0, p->c1, p->c2, alpha);
    }
    if (internal_msaa4x(ix+1, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1, p->c3, alpha)) {
        blend_single_color(color[1], p->c0, p->c1, p->c2, alpha);
    }
    if (internal_msaa4x(ix, iy+1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1, p->c3, alpha)) {
        blend_single_color(color[2], p->c0, p->c1, p->c2, alpha);
    }
    if (internal_msaa4x(ix+1, iy+1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1, p->c3, alpha)) {
        blend_single_color(color[3], p->c0, p->c1, p->c2, alpha);
    }
}

// render_rectangle_fill:
// render filled rectangle with border msaa4x interpolation off
static __device__ void render_rectangle_fill(int ix, int iy, RectangleCommand* p, uchar4 color[4]) {
    if (inbox_single_pixel(ix, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1)) {
        blend_single_color(color[0], p->c0, p->c1, p->c2, p->c3);
    }
    if (inbox_single_pixel(ix+1, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1)) {
        blend_single_color(color[1], p->c0, p->c1, p->c2, p->c3);
    }
    if (inbox_single_pixel(ix, iy+1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1)) {
        blend_single_color(color[2], p->c0, p->c1, p->c2, p->c3);
    }
    if (inbox_single_pixel(ix+1, iy+1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1)) {
        blend_single_color(color[3], p->c0, p->c1, p->c2, p->c3);
    }
}

// render_rectangle_border_msaa4x:
// render hollow rectangle with border msaa4x interpolation on
static __device__ void render_rectangle_border_msaa4x(int ix, int iy, RectangleCommand* p, uchar4 color[4]) {
    unsigned char alpha;
    if (internal_msaa4x(ix, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1, p->c3, alpha) &&
        external_msaa4x(ix, iy, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2, p->c3, alpha)
    ) {
        blend_single_color(color[0], p->c0, p->c1, p->c2, alpha);
    }
    if (internal_msaa4x(ix+1, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1, p->c3, alpha) &&
        external_msaa4x(ix+1, iy, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2, p->c3, alpha)
    ) {
        blend_single_color(color[1], p->c0, p->c1, p->c2, alpha);
    }
    if (internal_msaa4x(ix, iy+1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1, p->c3, alpha) &&
        external_msaa4x(ix, iy+1, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2, p->c3, alpha)
    ) {
        blend_single_color(color[2], p->c0, p->c1, p->c2, alpha);
    }
    if (internal_msaa4x(ix+1, iy+1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1, p->c3, alpha) &&
        external_msaa4x(ix+1, iy+1, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2, p->c3, alpha)
    ) {
        blend_single_color(color[3], p->c0, p->c1, p->c2, alpha);
    }
}

// render_rectangle_border:
// render hollow rectangle with border msaa4x interpolation off
static __device__ void render_rectangle_border(int ix, int iy, RectangleCommand* p, uchar4 color[4]) {
    if (!inbox_single_pixel(ix, iy, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2) &&
        inbox_single_pixel(ix, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1)
    ) {
        blend_single_color(color[0], p->c0, p->c1, p->c2, p->c3);
    }
    if (!inbox_single_pixel(ix+1, iy, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2) &&
        inbox_single_pixel(ix+1, iy, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1)
    ) {
        blend_single_color(color[1], p->c0, p->c1, p->c2, p->c3);
    }
    if (!inbox_single_pixel(ix, iy+1, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2) &&
        inbox_single_pixel(ix, iy+1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1)
    ) {
        blend_single_color(color[2], p->c0, p->c1, p->c2, p->c3);
    }
    if (!inbox_single_pixel(ix+1, iy+1, p->ax2, p->ay2, p->bx2, p->by2, p->cx2, p->cy2, p->dx2, p->dy2) &&
        inbox_single_pixel(ix+1, iy+1, p->ax1, p->ay1, p->bx1, p->by1, p->cx1, p->cy1, p->dx1, p->dy1)
    ) {
        blend_single_color(color[3], p->c0, p->c1, p->c2, p->c3);
    }
}

// render_circle_interpolation:
// render cicle with border interpolation
static __device__ void render_circle_interpolation(
    int ix, int iy, CircleCommand* p, uchar4 color[4]
) {
    float tr0 = sqrt((float)(ix - p->cx) * (ix - p->cx) + (iy - p->cy) * (iy - p->cy));
    float tr1 = sqrt((float)(ix + 1 - p->cx) * (ix + 1 - p->cx) + (iy - p->cy) * (iy - p->cy));
    float tr2 = sqrt((float)(ix - p->cx) * (ix - p->cx) + (iy + 1 - p->cy) * (iy + 1 - p->cy));
    float tr3 = sqrt((float)(ix + 1 - p->cx) * (ix + 1 - p->cx) + (iy + 1 - p->cy) * (iy + 1 - p->cy));

    int inner_boundsize = p->radius - p->thickness / 2;
    int external_boundsize = inner_boundsize + p->thickness;

    if (p->thickness < 0) {
        if (p->thickness == -1) {
            external_boundsize = p->radius;
        } else {
            external_boundsize = inner_boundsize;
        }
        inner_boundsize = 0;
    }

    unsigned char alpha0 = interpolation_fn(tr0, inner_boundsize, external_boundsize, 1, p->c3);
    unsigned char alpha1 = interpolation_fn(tr1, inner_boundsize, external_boundsize, 1, p->c3);
    unsigned char alpha2 = interpolation_fn(tr2, inner_boundsize, external_boundsize, 1, p->c3);
    unsigned char alpha3 = interpolation_fn(tr3, inner_boundsize, external_boundsize, 1, p->c3);

    if (alpha0){blend_single_color(color[0], p->c0, p->c1, p->c2, alpha0);}
    if (alpha1){blend_single_color(color[1], p->c0, p->c1, p->c2, alpha1);}
    if (alpha2){blend_single_color(color[2], p->c0, p->c1, p->c2, alpha2);}
    if (alpha3){blend_single_color(color[3], p->c0, p->c1, p->c2, alpha3);}
}

static __device__ void sample_pixel_bilinear(
    float* d_ptr, int x, int y, float sx, float sy, int width, int height, float threshold, unsigned char& a
) {
    float src_x = (x + 0.5f) * sx - 0.5f;
    float src_y = (y + 0.5f) * sy - 0.5f;
    int y_low  = floorf(src_y);
    int x_low  = floorf(src_x);
    int y_high = limit(y_low + 1, 0, height - 1);
    int x_high = limit(x_low + 1, 0, width - 1);
    y_low = limit(y_low, 0, height - 1);
    x_low = limit(x_low, 0, width - 1);

    int ly = rint((src_y - y_low) * INTER_RESIZE_COEF_SCALE);
    int lx = rint((src_x - x_low) * INTER_RESIZE_COEF_SCALE);
    int hy = INTER_RESIZE_COEF_SCALE - ly;
    int hx = INTER_RESIZE_COEF_SCALE - lx;

    uchar4 _scr;

    _scr.x = d_ptr[x_low + y_low * width] > threshold ? 127 : 0;
    _scr.y = d_ptr[x_high + y_low * width] > threshold ? 127 : 0;
    _scr.z = d_ptr[x_low + y_high * width] > threshold ? 127 : 0;
    _scr.w = d_ptr[x_high + y_high * width] > threshold ? 127 : 0;

    a = ( ((hy * ((hx * _scr.x + lx * _scr.y) >> 4)) >> 16) + ((ly * ((hx * _scr.z + lx * _scr.w) >> 4)) >> 16) + 2 )>>2;
}

static __device__ bool isRayIntersectsSegment(int p0, int p1, int s0, int s1, int e0, int e1) {
	if (s1 == e1)
		return false;
	if (s1 > p1 && e1 > p1)
		return false;
	if (s1 < p1 && e1 < p1)
		return false;
	if (s1 == p1 && e1 > p1)
		return false;
	if (e1 == p1 && s1 > p1)
		return false;
	if (s0 < p0 && e0 < p0)
		return false;
	int xseg = e0 - (e0 - s0) * (e1 - p1) / (e1 - s1);
	if (xseg < p0)
		return false;
	return true;
}

static __device__ void render_polyfill(
    int ix, int iy, PolyFillCommand* p, uchar4 color[4]
) {
    if (ix + 1 < p->bounding_left || iy + 1 < p->bounding_top || ix >= p->bounding_right || iy >= p->bounding_bottom)
        return;

	int sinsc[4] = { 0, 0, 0, 0 };
    for (int i=0; i<p->n_pts; i++)
    {
        if(i==0) {
			if (isRayIntersectsSegment(ix, iy, p->d_pts[0], p->d_pts[1], p->d_pts[p->n_pts * 2 - 2], p->d_pts[p->n_pts * 2 - 1])) sinsc[0] += 1;
			if (isRayIntersectsSegment(ix+1, iy, p->d_pts[0], p->d_pts[1], p->d_pts[p->n_pts * 2 - 2], p->d_pts[p->n_pts * 2 - 1])) sinsc[1] += 1;
			if (isRayIntersectsSegment(ix, iy+1, p->d_pts[0], p->d_pts[1], p->d_pts[p->n_pts * 2 - 2], p->d_pts[p->n_pts * 2 - 1])) sinsc[2] += 1;
			if (isRayIntersectsSegment(ix+1, iy+1, p->d_pts[0], p->d_pts[1], p->d_pts[p->n_pts * 2 - 2], p->d_pts[p->n_pts * 2 - 1])) sinsc[3] += 1;
        }
        else {
			if (isRayIntersectsSegment(ix, iy, p->d_pts[i * 2 - 2], p->d_pts[i * 2 - 1], p->d_pts[i * 2], p->d_pts[i * 2 + 1])) sinsc[0] += 1;
			if (isRayIntersectsSegment(ix+1, iy, p->d_pts[i * 2 - 2], p->d_pts[i * 2 - 1], p->d_pts[i * 2], p->d_pts[i * 2 + 1])) sinsc[1] += 1;
			if (isRayIntersectsSegment(ix, iy+1, p->d_pts[i * 2 - 2], p->d_pts[i * 2 - 1], p->d_pts[i * 2], p->d_pts[i * 2 + 1])) sinsc[2] += 1;
			if (isRayIntersectsSegment(ix+1, iy+1, p->d_pts[i * 2 - 2], p->d_pts[i * 2 - 1], p->d_pts[i * 2], p->d_pts[i * 2 + 1])) sinsc[3] += 1;
        }
    }

    if(sinsc[0] %2 !=0) {
        blend_single_color(color[0], p->c0, p->c1, p->c2, p->c3);
    }

    if(sinsc[1] %2 !=0) {
        blend_single_color(color[1], p->c0, p->c1, p->c2, p->c3);
    }

    if(sinsc[2] %2 !=0) {
        blend_single_color(color[2], p->c0, p->c1, p->c2, p->c3);
    }

    if(sinsc[3] %2 !=0) {
        blend_single_color(color[3], p->c0, p->c1, p->c2, p->c3);
    }
}

static __device__ void render_segment_bilinear(
    int ix, int iy, SegmentCommand* p, uchar4 color[4]
) {
    if (ix + 1 < p->bounding_left || iy + 1 < p->bounding_top || ix >= p->bounding_right || iy >= p->bounding_bottom)
        return;

    unsigned char alpha0 = ix   < p->bounding_left || iy < p->bounding_top   || ix >= p->bounding_right   || iy >= p->bounding_bottom ? 0 : 127;
    unsigned char alpha1 = ix+1 < p->bounding_left || iy < p->bounding_top   || ix+1 >= p->bounding_right || iy >= p->bounding_bottom ? 0 : 127;
    unsigned char alpha2 = ix   < p->bounding_left || iy+1 < p->bounding_top || ix >= p->bounding_right   || iy+1 >= p->bounding_bottom ? 0 : 127;
    unsigned char alpha3 = ix+1 < p->bounding_left || iy+1 < p->bounding_top || ix+1 >= p->bounding_right || iy+1 >= p->bounding_bottom ? 0 : 127;

    int fx = ix - p->bounding_left;
    int fy = iy - p->bounding_top;

    if(alpha0) {
        sample_pixel_bilinear(p->d_seg, fx, fy, p->scale_x, p->scale_y, p->seg_width, p->seg_height, p->seg_threshold, alpha0);
        blend_single_color(color[0], p->c0, p->c1, p->c2, alpha0);
    }

    if(alpha1) {
        sample_pixel_bilinear(p->d_seg, fx+1, fy, p->scale_x, p->scale_y, p->seg_width, p->seg_height, p->seg_threshold, alpha1);
        blend_single_color(color[1], p->c0, p->c1, p->c2, alpha1);
    }

    if(alpha2) {
        sample_pixel_bilinear(p->d_seg, fx, fy+1, p->scale_x, p->scale_y, p->seg_width, p->seg_height, p->seg_threshold, alpha2);
        blend_single_color(color[2], p->c0, p->c1, p->c2, alpha2);
    }

    if(alpha3) {
        sample_pixel_bilinear(p->d_seg, fx+1, fy+1, p->scale_x, p->scale_y, p->seg_width, p->seg_height, p->seg_threshold, alpha3);
        blend_single_color(color[3], p->c0, p->c1, p->c2, alpha3);
    }
}

static __device__ bool render_text(
    int ix, int iy, const TextLocation& location, const unsigned char* text_bitmap, int text_bitmap_width,
    uchar4 color[4], unsigned char& c0, unsigned char& c1, unsigned char& c2, unsigned char& a
) {
    if (ix + 1 < location.image_x || iy + 1 < location.image_y || ix >= location.image_x + location.text_w || iy >= location.image_y + location.text_h)
        return false;

    int fx  = ix - location.image_x;
    int fy  = iy - location.image_y;
    int bfx = fx + location.text_x;
    unsigned char alpha0 = fx < 0   || fy < 0   || fx >= location.text_w   || fy >= location.text_h   ? 0 : ((text_bitmap[fy * text_bitmap_width + bfx + 0] * (int)a) >> 8);
    unsigned char alpha1 = fx+1 < 0 || fy < 0   || fx+1 >= location.text_w || fy >= location.text_h   ? 0 : ((text_bitmap[fy * text_bitmap_width + bfx + 1] * (int)a) >> 8);
    unsigned char alpha2 = fx < 0   || fy+1 < 0 || fx >= location.text_w   || fy+1 >= location.text_h ? 0 : ((text_bitmap[(fy + 1) * text_bitmap_width + bfx + 0] * (int)a) >> 8);
    unsigned char alpha3 = fx+1 < 0 || fy+1 < 0 || fx+1 >= location.text_w || fy+1 >= location.text_h ? 0 : ((text_bitmap[(fy + 1) * text_bitmap_width + bfx + 1] * (int)a) >> 8);

    if (alpha0){blend_single_color(color[0], c0, c1, c2, alpha0);}
    if (alpha1){blend_single_color(color[1], c0, c1, c2, alpha1);}
    if (alpha2){blend_single_color(color[2], c0, c1, c2, alpha2);}
    if (alpha3){blend_single_color(color[3], c0, c1, c2, alpha3);}
    return true;
}

static __device__ void blend_nv12_bilinear(
    void* d_ptr0, void* d_ptr1, int x, int y, float sx, float sy, int width, int stride, int height, uchar4* color, unsigned char c3, bool block_linear
) {
    float src_x = (x + 0.5f) * sx - 0.5f;
    float src_y = (y + 0.5f) * sy - 0.5f;
    int y_low  = floorf(src_y);
    int x_low  = floorf(src_x);
    int y_high = limit(y_low + 1, 0, height - 1);
    int x_high = limit(x_low + 1, 0, width - 1);
    y_low = limit(y_low, 0, height - 1);
    x_low = limit(x_low, 0, width - 1);

    int ly = rint((src_y - y_low) * INTER_RESIZE_COEF_SCALE);
    int lx = rint((src_x - x_low) * INTER_RESIZE_COEF_SCALE);
    int hy = INTER_RESIZE_COEF_SCALE - ly;
    int hx = INTER_RESIZE_COEF_SCALE - lx;

    uchar4 _scr[5];

    if (block_linear) {
        _scr[0].x = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr0, x_low, y_low);
        _scr[1].x = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr0, x_high, y_low);
        _scr[2].x = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr0, x_low, y_high);
        _scr[3].x = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr0, x_high, y_high);

        _scr[0].y = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr1, 2 * (x_low >> 1), y_low >> 1);
        _scr[1].y = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr1, 2 * (x_high >> 1), y_low >> 1);
        _scr[2].y = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr1, 2 * (x_low >> 1), y_high >> 1);
        _scr[3].y = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr1, 2 * (x_high >> 1), y_high >> 1);

        _scr[0].z = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr1, 2 * (x_low >> 1) + 1, y_low >> 1);
        _scr[1].z = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr1, 2 * (x_high >> 1) + 1, y_low >> 1);
        _scr[2].z = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr1, 2 * (x_low >> 1) + 1, y_high >> 1);
        _scr[3].z = surf2Dread<unsigned char>((hipSurfaceObject_t)d_ptr1, 2 * (x_high >> 1) + 1, y_high >> 1);
    }
    else {
        _scr[0] = make_uchar4(((unsigned char *)d_ptr0)[y_low * stride + x_low], ((unsigned char *)d_ptr1)[(y_low >> 1)* stride + 2 * (x_low >> 1)], ((unsigned char *)d_ptr1)[(y_low >> 1)* stride + 2 * (x_low >> 1) + 1], c3);
        _scr[1] = make_uchar4(((unsigned char *)d_ptr0)[y_low * stride + x_high], ((unsigned char *)d_ptr1)[(y_low >> 1)* stride + 2 * (x_high >> 1)], ((unsigned char *)d_ptr1)[(y_low >> 1)* stride + 2 * (x_high >> 1) + 1], c3);
        _scr[2] = make_uchar4(((unsigned char *)d_ptr0)[y_high * stride + x_low], ((unsigned char *)d_ptr1)[(y_high >> 1)* stride + 2 * (x_low >> 1)], ((unsigned char *)d_ptr1)[(y_high >> 1)* stride + 2 * (x_low >> 1) + 1], c3);
        _scr[3] = make_uchar4(((unsigned char *)d_ptr0)[y_high * stride + x_high], ((unsigned char *)d_ptr1)[(y_high >> 1)* stride + 2 * (x_high >> 1)], ((unsigned char *)d_ptr1)[(y_high >> 1)* stride + 2 * (x_high >> 1) + 1], c3);
    }

    yuv2rgb(_scr[0].x, _scr[0].y, _scr[0].z, _scr[0].x, _scr[0].y, _scr[0].z);
    yuv2rgb(_scr[1].x, _scr[1].y, _scr[1].z, _scr[1].x, _scr[1].y, _scr[1].z);
    yuv2rgb(_scr[2].x, _scr[2].y, _scr[2].z, _scr[2].x, _scr[2].y, _scr[2].z);
    yuv2rgb(_scr[3].x, _scr[3].y, _scr[3].z, _scr[3].x, _scr[3].y, _scr[3].z);

    _scr[4].x = ( ((hy * ((hx * _scr[0].x + lx * _scr[1].x) >> 4)) >> 16) + ((ly * ((hx * _scr[2].x + lx * _scr[3].x) >> 4)) >> 16) + 2 )>>2;
    _scr[4].y = ( ((hy * ((hx * _scr[0].y + lx * _scr[1].y) >> 4)) >> 16) + ((ly * ((hx * _scr[2].y + lx * _scr[3].y) >> 4)) >> 16) + 2 )>>2;
    _scr[4].z = ( ((hy * ((hx * _scr[0].z + lx * _scr[1].z) >> 4)) >> 16) + ((ly * ((hx * _scr[2].z + lx * _scr[3].z) >> 4)) >> 16) + 2 )>>2;

    blend_single_color(color[0], _scr[4].x, _scr[4].y, _scr[4].z, c3);
}

// render_bl_nv12_src:
// render color from nv12 bl source image
static __device__ void render_nv12_src(
    int ix, int iy, NV12SourceCommand* p, uchar4 color[4]
) {
    if (ix + 1 < p->bounding_left || iy + 1 < p->bounding_top || ix >= p->bounding_right || iy >= p->bounding_bottom)
        return;

    unsigned char alpha0 = ix   < p->bounding_left || iy < p->bounding_top   || ix >= p->bounding_right   || iy >= p->bounding_bottom ? 0 : 127;
    unsigned char alpha1 = ix+1 < p->bounding_left || iy < p->bounding_top   || ix+1 >= p->bounding_right || iy >= p->bounding_bottom ? 0 : 127;
    unsigned char alpha2 = ix   < p->bounding_left || iy+1 < p->bounding_top || ix >= p->bounding_right   || iy+1 >= p->bounding_bottom ? 0 : 127;
    unsigned char alpha3 = ix+1 < p->bounding_left || iy+1 < p->bounding_top || ix+1 >= p->bounding_right || iy+1 >= p->bounding_bottom ? 0 : 127;

    int fx = ix - p->bounding_left;
    int fy = iy - p->bounding_top;

    if(alpha0) {
        blend_nv12_bilinear(p->d_src0, p->d_src1, fx, fy, p->scale_x, p->scale_y, p->src_width, p->src_stride, p->src_height, &color[0], p->c3, p->block_linear);
    }

    if(alpha1) {
        blend_nv12_bilinear(p->d_src0, p->d_src1, fx+1, fy, p->scale_x, p->scale_y, p->src_width, p->src_stride, p->src_height, &color[1], p->c3, p->block_linear);
    }

    if(alpha2) {
        blend_nv12_bilinear(p->d_src0, p->d_src1, fx, fy+1, p->scale_x, p->scale_y, p->src_width, p->src_stride, p->src_height, &color[2], p->c3, p->block_linear);
    }

    if(alpha3) {
        blend_nv12_bilinear(p->d_src0, p->d_src1, fx+1, fy+1, p->scale_x, p->scale_y, p->src_width, p->src_stride, p->src_height, &color[3], p->c3, p->block_linear);
    }
}

static __device__ void blend_rgba_bilinear(
    uint8_t* d_ptr, int x, int y, float sx, float sy, int width, int stride, int height, uchar4* color
) {
    float src_x = (x + 0.5f) * sx - 0.5f;
    float src_y = (y + 0.5f) * sy - 0.5f;
    int y_low  = floorf(src_y);
    int x_low  = floorf(src_x);
    int y_high = limit(y_low + 1, 0, height - 1);
    int x_high = limit(x_low + 1, 0, width - 1);
    y_low = limit(y_low, 0, height - 1);
    x_low = limit(x_low, 0, width - 1);

    int ly = rint((src_y - y_low) * INTER_RESIZE_COEF_SCALE);
    int lx = rint((src_x - x_low) * INTER_RESIZE_COEF_SCALE);
    int hy = INTER_RESIZE_COEF_SCALE - ly;
    int hx = INTER_RESIZE_COEF_SCALE - lx;

    uchar4 _scr[5];

    _scr[0] = *(uchar4 *)&d_ptr[4 * x_low + y_low * stride];
    _scr[1] = *(uchar4 *)&d_ptr[4 * x_high + y_low * stride];
    _scr[2] = *(uchar4 *)&d_ptr[4 * x_low + y_high * stride];
    _scr[3] = *(uchar4 *)&d_ptr[4 * x_high + y_high * stride];

    _scr[4].x = ( ((hy * ((hx * _scr[0].x + lx * _scr[1].x) >> 4)) >> 16) + ((ly * ((hx * _scr[2].x + lx * _scr[3].x) >> 4)) >> 16) + 2 )>>2;
    _scr[4].y = ( ((hy * ((hx * _scr[0].y + lx * _scr[1].y) >> 4)) >> 16) + ((ly * ((hx * _scr[2].y + lx * _scr[3].y) >> 4)) >> 16) + 2 )>>2;
    _scr[4].z = ( ((hy * ((hx * _scr[0].z + lx * _scr[1].z) >> 4)) >> 16) + ((ly * ((hx * _scr[2].z + lx * _scr[3].z) >> 4)) >> 16) + 2 )>>2;
    _scr[4].w = ( ((hy * ((hx * _scr[0].w + lx * _scr[1].w) >> 4)) >> 16) + ((ly * ((hx * _scr[2].w + lx * _scr[3].w) >> 4)) >> 16) + 2 )>>2;

    blend_single_color(color[0], _scr[4].x, _scr[4].y, _scr[4].z, _scr[4].w);
}

// render_rgba_src:
// render color from rgba source image
static __device__ void render_rgba_src(
    int ix, int iy, RGBASourceCommand* p, uchar4 color[4]
) {
    if (ix + 1 < p->bounding_left || iy + 1 < p->bounding_top || ix >= p->bounding_right || iy >= p->bounding_bottom)
        return;

    unsigned char alpha0 = ix   < p->bounding_left || iy < p->bounding_top   || ix >= p->bounding_right   || iy >= p->bounding_bottom ? 0 : 127;
    unsigned char alpha1 = ix+1 < p->bounding_left || iy < p->bounding_top   || ix+1 >= p->bounding_right || iy >= p->bounding_bottom ? 0 : 127;
    unsigned char alpha2 = ix   < p->bounding_left || iy+1 < p->bounding_top || ix >= p->bounding_right   || iy+1 >= p->bounding_bottom ? 0 : 127;
    unsigned char alpha3 = ix+1 < p->bounding_left || iy+1 < p->bounding_top || ix+1 >= p->bounding_right || iy+1 >= p->bounding_bottom ? 0 : 127;

    int fx = ix - p->bounding_left;
    int fy = iy - p->bounding_top;

    if(alpha0) {
        blend_rgba_bilinear((uint8_t *)p->d_src, fx, fy, p->scale_x, p->scale_y, p->src_width, p->src_stride, p->src_height, &color[0]);
    }

    if(alpha1) {
        blend_rgba_bilinear((uint8_t *)p->d_src, fx+1, fy, p->scale_x, p->scale_y, p->src_width, p->src_stride, p->src_height, &color[1]);
    }

    if(alpha2) {
        blend_rgba_bilinear((uint8_t *)p->d_src, fx, fy+1, p->scale_x, p->scale_y, p->src_width, p->src_stride, p->src_height, &color[2]);
    }

    if(alpha3) {
        blend_rgba_bilinear((uint8_t *)p->d_src, fx+1, fy+1, p->scale_x, p->scale_y, p->src_width, p->src_stride, p->src_height, &color[3]);
    }
}

template<ImageFormat format>
struct BlendingPixel{};

template<>
struct BlendingPixel<ImageFormat::RGBA>{
    static __device__ void call(
        const void* image0, const void* image1,
        int x, int y, int stride, uchar4 plot_colors[4]
    ) {
        for (int i = 0; i < 2; ++i) {
            unsigned char* p = ((unsigned char*)image0 + (y + i) * stride + x * 4);
            for (int j = 0; j < 2; ++j, p += 4) {
                uchar4& rcolor   = plot_colors[i * 2 + j];
                int foreground_alpha = rcolor.w;
                int background_alpha = p[3];
                int blend_alpha      = ((background_alpha * (255 - foreground_alpha)) >> 8) + foreground_alpha;
                p[0] = u8cast((((p[0] * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.x * foreground_alpha)) / blend_alpha);
                p[1] = u8cast((((p[1] * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.y * foreground_alpha)) / blend_alpha);
                p[2] = u8cast((((p[2] * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.z * foreground_alpha)) / blend_alpha);
                p[3] = blend_alpha;
            }
        }
    }
};

template<>
struct BlendingPixel<ImageFormat::RGB>{
    static __device__ void call(
        const void* image0, const void* image1,
        int x, int y, int stride, uchar4 plot_colors[4]
    ) {
        for (int i = 0; i < 2; ++i) {
            unsigned char* p = ((unsigned char*)image0 + (y + i) * stride + x * 3);
            for (int j = 0; j < 2; ++j, p += 3) {
                uchar4& rcolor   = plot_colors[i * 2 + j];
                int foreground_alpha = rcolor.w;
                int background_alpha = 255;
                int blend_alpha      = ((background_alpha * (255 - foreground_alpha)) >> 8) + foreground_alpha;
                p[0] = u8cast((((p[0] * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.x * foreground_alpha)) / blend_alpha);
                p[1] = u8cast((((p[1] * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.y * foreground_alpha)) / blend_alpha);
                p[2] = u8cast((((p[2] * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.z * foreground_alpha)) / blend_alpha);
            }
        }
    }
};

template<>
struct BlendingPixel<ImageFormat::BlockLinearNV12>{
    static __device__ void call(
        const void* image0, const void* image1,
        int x, int y, int stride, uchar4 plot_colors[4]
    ) {
        unsigned char img_y0 = surf2Dread<unsigned char>((hipSurfaceObject_t)image0, x + 0, y);
        unsigned char img_y1 = surf2Dread<unsigned char>((hipSurfaceObject_t)image0, x + 1, y);
        unsigned char img_y2 = surf2Dread<unsigned char>((hipSurfaceObject_t)image0, x + 0, y + 1);
        unsigned char img_y3 = surf2Dread<unsigned char>((hipSurfaceObject_t)image0, x + 1, y + 1);

        unsigned char img_u = surf2Dread<unsigned char>((hipSurfaceObject_t)image1, x,     y / 2);
        unsigned char img_v = surf2Dread<unsigned char>((hipSurfaceObject_t)image1, x + 1, y / 2);

        uchar3 rgb[4];
        yuv2rgb(img_y0, img_u, img_v, rgb[0].x, rgb[0].y, rgb[0].z);
        yuv2rgb(img_y1, img_u, img_v, rgb[1].x, rgb[1].y, rgb[1].z);
        yuv2rgb(img_y2, img_u, img_v, rgb[2].x, rgb[2].y, rgb[2].z);
        yuv2rgb(img_y3, img_u, img_v, rgb[3].x, rgb[3].y, rgb[3].z);

        for (int i = 0; i < 4; ++i) {
            uchar4& rcolor   = plot_colors[i];
            uchar3& lcolor   = rgb[i];
            int foreground_alpha = rcolor.w;
            int background_alpha = 255;
            int blend_alpha      = ((background_alpha * (255 - foreground_alpha)) >> 8) + foreground_alpha;
            lcolor.x = u8cast((((lcolor.x * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.x * foreground_alpha)) / blend_alpha);
            lcolor.y = u8cast((((lcolor.y * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.y * foreground_alpha)) / blend_alpha);
            lcolor.z = u8cast((((lcolor.z * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.z * foreground_alpha)) / blend_alpha);
        }

        uchar4 img_u4, img_v4;

        rgb2yuv(rgb[0].x, rgb[0].y, rgb[0].z, img_y0, img_u4.x, img_v4.x);
        rgb2yuv(rgb[1].x, rgb[1].y, rgb[1].z, img_y1, img_u4.y, img_v4.y);
        rgb2yuv(rgb[2].x, rgb[2].y, rgb[2].z, img_y2, img_u4.z, img_v4.z);
        rgb2yuv(rgb[3].x, rgb[3].y, rgb[3].z, img_y3, img_u4.w, img_v4.w);

        int meanu = (img_u4.x + img_u4.y + img_u4.z + img_u4.w) / 4;
        int meanv = (img_v4.x + img_v4.y + img_v4.z + img_v4.w) / 4;

        surf2Dwrite<unsigned char>(img_y0, (hipSurfaceObject_t)image0, x + 0, y);
        surf2Dwrite<unsigned char>(img_y1, (hipSurfaceObject_t)image0, x + 1, y);
        surf2Dwrite<unsigned char>(img_y2, (hipSurfaceObject_t)image0, x + 0, y + 1);
        surf2Dwrite<unsigned char>(img_y3, (hipSurfaceObject_t)image0, x + 1, y + 1);

        surf2Dwrite<unsigned char>(u8cast(meanu), (hipSurfaceObject_t)image1, x    , y / 2);
        surf2Dwrite<unsigned char>(u8cast(meanv), (hipSurfaceObject_t)image1, x + 1, y / 2);
    }
};

template<>
struct BlendingPixel<ImageFormat::PitchLinearNV12>{
    static __device__ void call(
        const void* image0, const void* image1,
        int x, int y, int stride, uchar4 plot_colors[4]
    ) {
        unsigned char& img_y0 = *((unsigned char*)image0 + y * stride + x + 0);
        unsigned char& img_y1 = *((unsigned char*)image0 + y * stride + x + 1);
        unsigned char& img_y2 = *((unsigned char*)image0 + (y+1) * stride + x + 0);
        unsigned char& img_y3 = *((unsigned char*)image0 + (y+1) * stride + x + 1);

        unsigned char* img_uv_ptr = (unsigned char*)image1 + (y / 2) * stride + x;
        unsigned char& img_u = img_uv_ptr[0];
        unsigned char& img_v = img_uv_ptr[1];

        uchar3 rgb[4];
        yuv2rgb(img_y0, img_u, img_v, rgb[0].x, rgb[0].y, rgb[0].z);
        yuv2rgb(img_y1, img_u, img_v, rgb[1].x, rgb[1].y, rgb[1].z);
        yuv2rgb(img_y2, img_u, img_v, rgb[2].x, rgb[2].y, rgb[2].z);
        yuv2rgb(img_y3, img_u, img_v, rgb[3].x, rgb[3].y, rgb[3].z);

        for (int i = 0; i < 4; ++i) {
            uchar4& rcolor   = plot_colors[i];
            uchar3& lcolor   = rgb[i];
            int foreground_alpha = rcolor.w;
            int background_alpha = 255;
            int blend_alpha      = ((background_alpha * (255 - foreground_alpha)) >> 8) + foreground_alpha;
            lcolor.x = u8cast((((lcolor.x * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.x * foreground_alpha)) / blend_alpha);
            lcolor.y = u8cast((((lcolor.y * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.y * foreground_alpha)) / blend_alpha);
            lcolor.z = u8cast((((lcolor.z * background_alpha * (255 - foreground_alpha))>>8) + (rcolor.z * foreground_alpha)) / blend_alpha);
        }

        uchar4 img_u4, img_v4;

        rgb2yuv(rgb[0].x, rgb[0].y, rgb[0].z, img_y0, img_u4.x, img_v4.x);
        rgb2yuv(rgb[1].x, rgb[1].y, rgb[1].z, img_y1, img_u4.y, img_v4.y);
        rgb2yuv(rgb[2].x, rgb[2].y, rgb[2].z, img_y2, img_u4.z, img_v4.z);
        rgb2yuv(rgb[3].x, rgb[3].y, rgb[3].z, img_y3, img_u4.w, img_v4.w);

        int meanu = (img_u4.x + img_u4.y + img_u4.z + img_u4.w) / 4;
        int meanv = (img_v4.x + img_v4.y + img_v4.z + img_v4.w) / 4;

        img_u = u8cast(meanu);
        img_v = u8cast(meanv);
    }
};

template<bool have_rotate_msaa>
static __device__ void do_rectangle(RectangleCommand* cmd, int ix, int iy, uchar4 context_color[4]);

template<>
__device__ void do_rectangle<true>(RectangleCommand* cmd, int ix, int iy, uchar4 context_color[4]) {
    if (cmd->thickness == -1) {
        if (cmd->interpolation) {
            render_rectangle_fill_msaa4x(ix, iy, cmd, context_color);
        } else {
            render_rectangle_fill(ix, iy, cmd, context_color);
        }
    } else {
        if (cmd->interpolation) {
            render_rectangle_border_msaa4x(ix, iy, cmd, context_color);
        } else {
            render_rectangle_border(ix, iy, cmd, context_color);
        }
    }
}

template<>
__device__ void do_rectangle<false>(RectangleCommand* cmd, int ix, int iy, uchar4 context_color[4]) {
    if (cmd->thickness == -1) {
        render_rectangle_fill(ix, iy, cmd, context_color);
    } else {
        render_rectangle_border(ix, iy, cmd, context_color);
    }
}

// render_elements_kernel:
// main entry for launching render CUDA kernel
template<ImageFormat format, bool have_rotate_msaa>
static __global__ void render_elements_kernel(
    int bx, int by,
    const TextLocation* text_locations, const unsigned char* text_bitmap, int text_bitmap_width, const int* line_location_base,
    const unsigned char* commands, const int* command_offsets, int num_command,
    const void* image0, const void* image1,
    int image_width, int stride, int image_height
) {
    int ix = ((blockDim.x * blockIdx.x + threadIdx.x) << 1) + bx;
    int iy = ((blockDim.y * blockIdx.y + threadIdx.y) << 1) + by;
    if (ix < 0 || iy < 0 || ix >= image_width - 1 || iy >= image_height - 1)
        return;

    int itext_line          = 0;
    uchar4 context_color[4] = {0};
    for (int i = 0; i < num_command; ++i) {
        cuOSDContextCommand* pcommand = (cuOSDContextCommand*)(commands + command_offsets[i]);

        // because there is four pixel to operator
        if (ix + 1 < pcommand->bounding_left || ix > pcommand->bounding_right ||
           iy + 1 < pcommand->bounding_top || iy > pcommand->bounding_bottom) {

            if (pcommand->type == CommandType::Text)
                itext_line++;
            continue;
        }

        switch(pcommand->type) {
            case CommandType::Rectangle:{
                RectangleCommand* rect_cmd = (RectangleCommand*)pcommand;
                do_rectangle<have_rotate_msaa>(rect_cmd, ix, iy, context_color);
                break;
            }
            case CommandType::Text:{
                int ilocation_begin = line_location_base[itext_line];
                int ilocation_end   = line_location_base[itext_line + 1];
                itext_line++;

                for (int j = ilocation_begin; j < ilocation_end; ++j) {
                    bool hit = render_text(
                        ix, iy, text_locations[j], text_bitmap, text_bitmap_width,
                        context_color, pcommand->c0, pcommand->c1, pcommand->c2, pcommand->c3
                    );
                    if (hit) break;
                }
                break;
            }
            case CommandType::Circle:{
                CircleCommand* circle_cmd = (CircleCommand*)pcommand;
                render_circle_interpolation(ix, iy, circle_cmd, context_color);
                break;
            }
            case CommandType::Segment:{
                SegmentCommand* seg_cmd = (SegmentCommand*)pcommand;
                render_segment_bilinear(ix, iy, seg_cmd, context_color);
                break;
            }
            case CommandType::PolyFill:{
                PolyFillCommand* poly_cmd = (PolyFillCommand*)pcommand;
                render_polyfill(ix, iy, poly_cmd, context_color);
                break;
            }
            case CommandType::RGBASource:{
                RGBASourceCommand* rgba_src_cmd = (RGBASourceCommand*)pcommand;
                render_rgba_src(ix, iy, rgba_src_cmd, context_color);
                break;
            }
            case CommandType::NV12Source:{
                NV12SourceCommand* nv12_src_cmd = (NV12SourceCommand*)pcommand;
                render_nv12_src(ix, iy, nv12_src_cmd, context_color);
                break;
            }
        }
    }

    if (context_color[0].w == 0 && context_color[1].w == 0 && context_color[2].w == 0 && context_color[3].w == 0)
        return;

    BlendingPixel<format>::call(image0, image1, ix, iy, stride, context_color);
}

template<ImageFormat format>
static __device__ void __forceinline__ load_pixel(
    const void* luma, const void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
);

// BL sample pixel implmentation
template<>
__device__ void __forceinline__ load_pixel<ImageFormat::BlockLinearNV12>(
    const void* luma, const void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    uint8_t yv = surf2Dread<uint8_t>((hipTextureObject_t)luma,   x,          y    );
    uint8_t uv = surf2Dread<uint8_t>((hipTextureObject_t)chroma, down_x + 0, y / 2);
    uint8_t vv = surf2Dread<uint8_t>((hipTextureObject_t)chroma, down_x + 1, y / 2);
    yuv2rgb(yv, uv, vv, r, g, b);
}

// PL sample pixel implmentation
template<>
__device__ void __forceinline__ load_pixel<ImageFormat::PitchLinearNV12>(
    const void* luma, const void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    uint8_t yv = *((const unsigned char*)luma + y * stride + x);
    uint8_t uv = *((const unsigned char*)chroma + (y / 2) * stride + down_x + 0);
    uint8_t vv = *((const unsigned char*)chroma + (y / 2) * stride + down_x + 1);
    yuv2rgb(yv, uv, vv, r, g, b);
}

template<>
__device__ void __forceinline__ load_pixel<ImageFormat::RGB>(
    const void* luma, const void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    uchar3 pixel = *(uchar3*)((const unsigned char*)luma + y * stride + x * 3);
    r = pixel.x;
    g = pixel.y;
    b = pixel.z;
}

template<>
__device__ void __forceinline__ load_pixel<ImageFormat::RGBA>(
    const void* luma, const void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    uchar4 pixel = *(uchar4*)((const unsigned char*)luma + y * stride + x * 4);
    r = pixel.x;
    g = pixel.y;
    b = pixel.z;
}

template<ImageFormat format>
static __device__ void __forceinline__ save_pixel(
    void* luma, void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
);

// BL sample pixel implmentation
template<>
__device__ void __forceinline__ save_pixel<ImageFormat::BlockLinearNV12>(
    void* luma, void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    uint8_t vy, vu, vv;
    rgb2yuv(r, g, b, vy, vu, vv);
    surf2Dwrite<uint8_t>(vy, (hipTextureObject_t)luma,   x,          y    );
    surf2Dwrite<uint8_t>(vu, (hipTextureObject_t)chroma, down_x + 0, y / 2);
    surf2Dwrite<uint8_t>(vv, (hipTextureObject_t)chroma, down_x + 1, y / 2);
}

// PL sample pixel implmentation
template<>
__device__ void __forceinline__ save_pixel<ImageFormat::PitchLinearNV12>(
    void* luma, void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    uint8_t vy, vu, vv;
    rgb2yuv(r, g, b, vy, vu, vv);
    *((unsigned char*)luma + y * stride + x) = vy;
    *((unsigned char*)chroma + (y / 2) * stride + down_x + 0) = vu;
    *((unsigned char*)chroma + (y / 2) * stride + down_x + 1) = vv;
}

template<>
__device__ void __forceinline__ save_pixel<ImageFormat::RGB>(
    void* luma, void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    *(uchar3*)((const unsigned char*)luma + y * stride + x * 3) = make_uchar3(r, g, b);
}

template<>
__device__ void __forceinline__ save_pixel<ImageFormat::RGBA>(
    void* luma, void* chroma,
    int x, int y, int down_x, int width, int stride, uint8_t& r, uint8_t& g, uint8_t& b
){
    *(uchar4*)((const unsigned char*)luma + y * stride + x * 4) = make_uchar4(r, g, b, 255);
}

template<ImageFormat format>
static __global__ void render_blur_kernel(
    const BoxBlurCommand* commands, int num_command,
    void* image0, void* image1,
    int image_width, int stride, int image_height
) {
    __shared__ uchar3 crop[32][32];
    int ix = threadIdx.x;
    int iy = threadIdx.y;
    const BoxBlurCommand& box = commands[blockIdx.x];
    
    int boxwidth  = box.bounding_right  - box.bounding_left;
    int boxheight = box.bounding_bottom - box.bounding_top;
    int sx = limit((int)(ix / 32.0f * (float)boxwidth + 0.5f + box.bounding_left), 0, image_width);
    int sy = limit((int)(iy / 32.0f * (float)boxheight + 0.5f + box.bounding_top), 0, image_height);
    auto& pix = crop[iy][ix];
    load_pixel<format>(image0, image1, sx, sy, round_down2(sx), image_width, stride, pix.x, pix.y, pix.z);
    __syncthreads();

    uint3 color = make_uint3(0, 0, 0);
    int n = 0;
    for(int i = -box.kernel_size / 2; i <= box.kernel_size / 2; ++i){
        for(int j = -box.kernel_size / 2; j <= box.kernel_size / 2; ++j){
            int u = i + iy;
            int v = j + ix;
            if(u >= 0 && u < 32 && v >= 0 && v < 32){
                auto& c = crop[u][v];
                color.x += c.x;
                color.y += c.y;
                color.z += c.z;
                n++;
            }
        }
    }
    __syncthreads();
    crop[iy][ix] = make_uchar3(color.x / n, color.y / n, color.z / n);
    __syncthreads();

    int gap_width  = (boxwidth  + 31) / 32;
    int gap_height = (boxheight + 31) / 32;
    for(int i = 0; i < gap_height; ++i){
        for(int j = 0; j < gap_width; ++j){
            int fx = ix * gap_width + j + box.bounding_left;
            int fy = iy * gap_height + i + box.bounding_top;
            if(fx >= 0 && fx < image_width && fy >= 0 && fy < image_height){
                int sx = (ix * gap_width + j) / (float)boxwidth * 32;
                int sy = (iy * gap_height + i) / (float)boxheight * 32;
                if(sx < 32 && sy < 32){
                    auto& pix = crop[sy][sx];
                    // *(uchar3*)((char*)image0 + fy * stride + fx * 3) = crop[sy][sx];
                    save_pixel<format>(image0, image1, fx, fy, round_down2(sx), image_width, stride, pix.x, pix.y, pix.z);
                }
            }
        }
    }
}

typedef void(*cuosd_launch_kernel_impl_fptr)(
    void* image_data0, void* image_data1, int width, int stride, int height,
    const TextLocation* text_location, const unsigned char* text_bitmap, int text_bitmap_width, const int* line_location_base,
    const unsigned char* commands, const int* commands_offset, int num_commands,
    int bounding_left, int bounding_top, int bounding_right, int bounding_bottom,
    void* _stream
);

typedef void(*cuosd_launch_blur_kernel_impl_fptr)(
    void* image_data0, void* image_data1, int width, int stride, int height,
    const BoxBlurCommand* commands, int num_commands,
    void* _stream
);

template<ImageFormat format, bool have_rotate_msaa>
static void cuosd_launch_kernel_impl(
    void* image_data0, void* image_data1, int width, int stride, int height,
    const TextLocation* text_location, const unsigned char* text_bitmap, int text_bitmap_width, const int* line_location_base,
    const unsigned char* commands, const int* commands_offset, int num_commands,
    int bounding_left, int bounding_top, int bounding_right, int bounding_bottom,
    void* _stream
) {
    bounding_left   = max(min(bounding_left, width-1),    0);
    bounding_top    = max(min(bounding_top, height-1),    0);
    bounding_right  = max(min(bounding_right, width-1),   0);
    bounding_bottom = max(min(bounding_bottom, height-1), 0);

    bounding_left = round_down2(bounding_left);
    bounding_top  = round_down2(bounding_top);

    int bounding_width  = bounding_right - bounding_left + 1;
    int bounding_height = bounding_bottom - bounding_top + 1;
    if (bounding_width < 1 || bounding_height < 1) {
        CUOSD_PRINT_W("Please check if there is anything to draw, or cuosd_apply has been called\n");
        return;
    }

    hipStream_t stream = (hipStream_t)_stream;
    dim3 block(16, 8);
    dim3 grid(((bounding_width+1) / 2 + block.x - 1) / block.x, ((bounding_height+1) / 2 + block.y - 1) / block.y);
    render_elements_kernel<format, have_rotate_msaa> <<<grid, block, 0, stream>>>(
        bounding_left, bounding_top,
        text_location, text_bitmap, text_bitmap_width, line_location_base,
        commands,  commands_offset, num_commands,
        image_data0, image_data1, width, stride, height
    );
    hipError_t code = hipPeekAtLastError();
    if(code != hipSuccess){
        CUOSD_PRINT_E("Launch kernel (render_elements_kernel) failed, code = %d", static_cast<int>(code));
    }
}

template<ImageFormat format>
static void cuosd_launch_blur_kernel_impl(
    void* image_data0, void* image_data1, int width, int stride, int height,
    const BoxBlurCommand* commands, int num_commands,
    void* _stream
) {
    if (num_commands < 1) {
        CUOSD_PRINT_W("Please check if there is anything to draw, or cuosd_apply has been called\n");
        return;
    }

    hipStream_t stream = (hipStream_t)_stream;
    dim3 block(32, 32);
    dim3 grid(num_commands);
    render_blur_kernel<format> <<<grid, block, 0, stream>>>(
        commands, num_commands,
        image_data0, image_data1, width, stride, height
    );

    hipError_t code = hipPeekAtLastError();
    if(code != hipSuccess){
        CUOSD_PRINT_E("Launch kernel (render_blur_kernel) failed, code = %d", static_cast<int>(code));
    }
}

void cuosd_launch_kernel(
    void* image_data0, void* image_data1, int width, int stride, int height, ImageFormat format,
    const TextLocation* text_location, const unsigned char* text_bitmap, int text_bitmap_width, const int* line_location_base,
    const unsigned char* commands, const int* commands_offset, int num_commands,
    int bounding_left, int bounding_top, int bounding_right, int bounding_bottom,
    bool have_rotate_msaa, const unsigned char* blur_commands, int num_blur_commands,
    void* _stream
) { 
    if(num_blur_commands > 0){
        const static cuosd_launch_blur_kernel_impl_fptr func_list[] = {
            cuosd_launch_blur_kernel_impl<ImageFormat::RGB>,
            cuosd_launch_blur_kernel_impl<ImageFormat::RGBA>,
            cuosd_launch_blur_kernel_impl<ImageFormat::BlockLinearNV12>,
            cuosd_launch_blur_kernel_impl<ImageFormat::PitchLinearNV12>
        };

        int index = (int)format - 1;
        if (index < 0 || index >= (int)sizeof(func_list) / (int)sizeof(func_list[0])) {
            CUOSD_PRINT_E("Unsupported configure %d\n", (int)index);
            return;
        }

        func_list[index](
            image_data0, image_data1, width, stride, height, 
            (BoxBlurCommand*)blur_commands, num_blur_commands,
            _stream
        );
    }

    if(num_commands > 0){
        const static cuosd_launch_kernel_impl_fptr func_list[] = {
            cuosd_launch_kernel_impl<ImageFormat::RGB, false>,
            cuosd_launch_kernel_impl<ImageFormat::RGBA, false>,
            cuosd_launch_kernel_impl<ImageFormat::BlockLinearNV12, false>,
            cuosd_launch_kernel_impl<ImageFormat::PitchLinearNV12, false>,

            cuosd_launch_kernel_impl<ImageFormat::RGB, true>,
            cuosd_launch_kernel_impl<ImageFormat::RGBA, true>,
            cuosd_launch_kernel_impl<ImageFormat::BlockLinearNV12, true>,
            cuosd_launch_kernel_impl<ImageFormat::PitchLinearNV12, true>,
        };

        int index = (int)(have_rotate_msaa) * 4 + (int)format - 1;
        if (index < 0 || index >= (int)sizeof(func_list) / (int)sizeof(func_list[0])) {
            CUOSD_PRINT_E("Unsupported configure %d\n", (int)index);
            return;
        }

        func_list[index](
            image_data0, image_data1, width, stride, height, text_location, text_bitmap, text_bitmap_width, line_location_base,
            commands, commands_offset, num_commands,
            bounding_left, bounding_top, bounding_right, bounding_bottom,
            _stream
        );
    }
}