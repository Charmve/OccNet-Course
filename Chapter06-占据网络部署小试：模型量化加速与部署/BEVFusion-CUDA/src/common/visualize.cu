#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */
 
#include <hip/hip_fp16.h>
#include <cuosd.h>
#include <math.h>
#include <string.h>

#include <algorithm>
#include <vector>

#include "dtype.hpp"
#include "launch.cuh"
#include "tensor.hpp"
#include "visualize.hpp"

namespace nv {

#define UseFont "tool/simhei.ttf"
#define MaxDistance 50
#define INTER_RESIZE_COEF_BITS 11
#define INTER_RESIZE_COEF_SCALE (1 << INTER_RESIZE_COEF_BITS)
#define CAST_BITS (INTER_RESIZE_COEF_BITS << 1)

typedef std::tuple<std::vector<nvtype::Float2>, int, float, float> Box3DInfo;
std::vector<Box3DInfo> transformation_predictions(const nvtype::Float4* viewport_4x4,
                                                  const std::vector<Prediction>& predictions) {
  if (predictions.empty()) return {};

  const int number_of_corner = 8;
  std::vector<Box3DInfo> output;
  output.reserve(predictions.size());

  // 8 x 3
  const nvtype::Float3 offset_of_corners[number_of_corner] = {{-1, -1, -1}, {+1, -1, -1}, {+1, +1, -1}, {-1, +1, -1},
                                                              {-1, -1, +1}, {+1, -1, +1}, {+1, +1, +1}, {-1, +1, +1}};

  for (size_t idx_predict = 0; idx_predict < predictions.size(); ++idx_predict) {
    auto& item = predictions[idx_predict];
    float cos_rotation = cos(item.z_rotation);
    float sin_rotation = sin(item.z_rotation);

    std::vector<nvtype::Float2> box3d;
    box3d.reserve(number_of_corner);

    nvtype::Float4 row0 = viewport_4x4[0];
    nvtype::Float4 row1 = viewport_4x4[1];
    nvtype::Float4 row2 = viewport_4x4[2];
    float zdepth = item.position.x * row2.x + item.position.y * row2.y + item.position.z * row2.z + row2.w;

    for (int idx_corner = 0; idx_corner < number_of_corner; ++idx_corner) {
      auto& offset = offset_of_corners[idx_corner];
      nvtype::Float3 corner;
      nvtype::Float3 std_corner;
      std_corner.x = item.size.w * offset.x * 0.5f;
      std_corner.y = item.size.l * offset.y * 0.5f;
      std_corner.z = item.size.h * offset.z * 0.5f;

      corner.x = item.position.x + std_corner.x * cos_rotation + std_corner.y * sin_rotation;
      corner.y = item.position.y + std_corner.x * -sin_rotation + std_corner.y * cos_rotation;
      corner.z = item.position.z + std_corner.z;

      float image_x = corner.x * row0.x + corner.y * row0.y + corner.z * row0.z + row0.w;
      float image_y = corner.x * row1.x + corner.y * row1.y + corner.z * row1.z + row1.w;
      float weight = corner.x * row2.x + corner.y * row2.y + corner.z * row2.z + row2.w;

      if (image_x <= 0 || image_y <= 0 || weight <= 0) {
        break;
      }

      weight = std::max(1e-5f, std::min(1e5f, weight));
      box3d.emplace_back(image_x / weight, image_y / weight);
    }

    if (box3d.size() != number_of_corner) continue;

    output.emplace_back(box3d, item.id, item.score, zdepth);
  }

  std::sort(output.begin(), output.end(), [](const Box3DInfo& a, const Box3DInfo& b) { return std::get<3>(a) > std::get<3>(b); });
  return output;
}

class ImageArtistImplement : public ImageArtist {
 public:
  virtual ~ImageArtistImplement() {
    if (cuosd_) cuosd_context_destroy(cuosd_);
  }

  bool init(const ImageArtistParameter& param) {
    param_ = param;
    if (param_.classes.empty()) {
      // printf("Use default nuscenes classes configuration.\n");
      param_.classes = {{"car", 255, 158, 0},        {"truck", 255, 99, 71},   {"construction_vehicle", 233, 150, 70},
                        {"bus", 255, 69, 0},         {"trailer", 255, 140, 0}, {"barrier", 112, 128, 144},
                        {"motorcycle", 255, 61, 99}, {"bicycle", 220, 20, 60}, {"pedestrian", 0, 0, 230},
                        {"traffic_cone", 47, 79, 79}};
    }
    cuosd_ = cuosd_context_create();
    return cuosd_ != nullptr;
  }

  virtual void draw_prediction(int camera_index, const std::vector<Prediction>& predictions, bool flipx) override {
    auto points = transformation_predictions(this->param_.viewport_nx4x4.data() + camera_index * 4, predictions);
    size_t num = points.size();
    for (size_t i = 0; i < num; ++i) {
      auto& item = points[i];
      auto& corners = std::get<0>(item);
      auto label = std::get<1>(item);
      auto score = std::get<2>(item);
      const int idx_of_line[][2] = {
          {0, 1}, {1, 2}, {2, 3}, {3, 0}, {4, 5}, {5, 6}, {6, 7}, {7, 4}, {0, 4}, {1, 5}, {2, 6}, {3, 7},
      };

      NameAndColor* name_color = &default_name_color_;
      if (label >= 0 && label < static_cast<int>(param_.classes.size())) {
        name_color = &param_.classes[label];
      }

      float size = std::sqrt(std::pow(corners[6].x - corners[0].x, 2) + std::pow(corners[6].y - corners[0].y, 2));
      float minx = param_.image_width;
      float miny = param_.image_height;
      for (size_t ioff = 0; ioff < sizeof(idx_of_line) / sizeof(idx_of_line[0]); ++ioff) {
        auto p0 = corners[idx_of_line[ioff][0]];
        auto p1 = corners[idx_of_line[ioff][1]];
        if (flipx) {
          p0.x = param_.image_width - p0.x - 1;
          p1.x = param_.image_width - p1.x - 1;
        }
        minx = std::min(minx, std::min(p0.x, p1.x));
        miny = std::min(miny, std::min(p0.y, p1.y));
        cuosd_draw_line(cuosd_, p0.x, p0.y, p1.x, p1.y, 5, {name_color->r, name_color->g, name_color->b, 255});
      }

      size = std::max(size * 0.06f, 8.0f);
      auto title = nv::format("%s %.2f", name_color->name.c_str(), score);
      cuosd_draw_text(cuosd_, title.c_str(), size, UseFont, minx, miny, {name_color->r, name_color->g, name_color->b, 255},
                      {255, 255, 255, 200});
    }
  }

  virtual void apply(unsigned char* image_rgb_device, void* stream) override {
    cuosd_apply(cuosd_, image_rgb_device, nullptr, param_.image_width, param_.image_stride, param_.image_height,
                cuOSDImageFormat::RGB, stream);
  }

 private:
  cuOSDContext_t cuosd_ = nullptr;
  ImageArtistParameter param_;
  NameAndColor default_name_color_{"Unknow", 0, 0, 0};
};

std::shared_ptr<ImageArtist> create_image_artist(const ImageArtistParameter& param) {
  std::shared_ptr<ImageArtistImplement> instance(new ImageArtistImplement());
  if (!instance->init(param)) {
    printf("Failed to create ImageArtist\n");
    instance.reset();
  }
  return instance;
}

typedef struct {
  half val[5];
} half5;

template <typename _T>
static __host__ __device__ _T limit(_T value, _T amin, _T amax) {
  return value < amin ? amin : (value > amax ? amax : value);
}

static __global__ void draw_point_to(unsigned int num, const half5* points, float4* view_port, unsigned char* image,
                                     int image_width, int stride, int image_height) {
  unsigned int idx = cuda_linear_index;
  if (idx >= num) return;

  half5 point = points[idx];
  float px = point.val[0];
  float py = point.val[1];
  float pz = point.val[2];
  float reflection = point.val[3];
  float indensity = point.val[4];

  float4 r0 = view_port[0];
  float4 r1 = view_port[1];
  float4 r2 = view_port[2];
  float x = px * r0.x + py * r0.y + pz * r0.z + r0.w;
  float y = px * r1.x + py * r1.y + pz * r1.z + r1.w;
  float w = px * r2.x + py * r2.y + pz * r2.z + r2.w;

  if (w <= 0) return;

  x = x / w;
  y = y / w;

  if (x < 0 || x >= image_width || y < 0 || y >= image_height) {
    return;
  }

  int ix = static_cast<int>(x);
  int iy = static_cast<int>(y);
  float alpha = limit((pz + 5.0f) / 8.0f, 0.35f, 1.0f);
  unsigned char gray = limit(alpha * 255, 0.0f, 255.0f);
  *(uchar3*)&image[iy * stride + ix * 3] = make_uchar3(gray, gray, gray);
}

static std::vector<nvtype::Float4> rodrigues_rotation(float radian, const std::vector<float>& axis){
  /*
     Rodrigues Rotation
  */
  std::vector<nvtype::Float4> output(4);
  memset(&output[0], 0, output.size() * sizeof(nvtype::Float4));

  float nx = axis[0];
  float ny = axis[1];
  float nz = axis[2];
  float cos_val = cos(radian);
  float sin_val = sin(radian);
  output[3].w = 1;

  float a = 1 - cos_val;
  float identity[3][3] = {
    {1, 0, 0},
    {0, 1, 0},
    {0, 0, 1},
  };

  float M[3][3] = {
    {0, -nz, ny},
    {nz, 0, -nx},
    {-ny, nx, 0}
  };

  for(int i = 0; i < 3; ++i){
    for(int j = 0; j < 3; ++j){
      ((float*)&output[i])[j] = cos_val * identity[i][j] + a * axis[i] * axis[j] + sin_val * M[i][j];
    }
  }
  return output;
}

std::vector<nvtype::Float4> matmul(const std::vector<nvtype::Float4>& a, const std::vector<nvtype::Float4>& b){
  std::vector<nvtype::Float4> c(a.size());
  memset(&c[0], 0, c.size() * sizeof(nvtype::Float4));

  for(size_t m = 0; m < a.size(); ++m){
    auto& ra = a[m];
    auto& rc = c[m];
    for(size_t n = 0; n < b.size(); ++n){
      for(size_t k = 0; k < 4; ++k){
        auto& rb = b[k];
        ((float*)&rc)[n] += ((float*)&ra)[k] * ((float*)&rb)[n];
      }
    }
  }
  return c;
}

struct BEVArtistDrawPointCommand {
  const nvtype::half* points_device;
  unsigned int number_of_points;
};

class BEVArtistImplement : public BEVArtist {
 public:
  virtual ~BEVArtistImplement() {
    if (transform_matrix_device_) checkRuntime(hipFree(transform_matrix_device_));
    if (cuosd_) cuosd_context_destroy(cuosd_);
  }

  bool init(const BEVArtistParameter& param) {
    param_ = param;
    if (param_.classes.empty()) {
      // printf("Use default nuscenes classes configuration.\n");
      param_.classes = {{"car", 255, 158, 0},        {"truck", 255, 99, 71},   {"construction_vehicle", 233, 150, 70},
                        {"bus", 255, 69, 0},         {"trailer", 255, 140, 0}, {"barrier", 112, 128, 144},
                        {"motorcycle", 255, 61, 99}, {"bicycle", 220, 20, 60}, {"pedestrian", 0, 0, 230},
                        {"traffic_cone", 47, 79, 79}};
    }

    std::vector<nvtype::Float4> lidar2image = {{param_.norm_size / MaxDistance, 0, 0, param_.cx},
                     {0, -param_.norm_size / MaxDistance, 0, param_.cy},
                     {0, 0, 0, 1},
                     {0, 0, 0, 1}};

    transform_matrix_.resize(4);
    memset(&transform_matrix_[0], 0, sizeof(nvtype::Float4) * transform_matrix_.size());

    auto rotation_x = rodrigues_rotation(param.rotate_x / 180.0f * 3.141592653f, {1, 0, 0});
    auto rotation_z = rodrigues_rotation(10.0f / 180.0f * 3.141592653f, {0, 0, 1});
    transform_matrix_ = matmul(lidar2image, matmul(rotation_x, rotation_z));

    checkRuntime(hipMalloc(&transform_matrix_device_, sizeof(nvtype::Float4) * transform_matrix_.size()));
    checkRuntime(hipMemcpy(transform_matrix_device_, transform_matrix_.data(), sizeof(nvtype::Float4) * transform_matrix_.size(),
                            hipMemcpyHostToDevice));
    cuosd_ = cuosd_context_create();
    return cuosd_ != nullptr;
  }

  virtual void draw_lidar_points(const nvtype::half* points_device, unsigned int number_of_points) override {
    draw_point_cmds_.emplace_back(BEVArtistDrawPointCommand{points_device, number_of_points});
  }

  virtual void draw_ego() override {
    Prediction ego;
    ego.position.x = 0;
    ego.position.y = 0;
    ego.position.z = 0;
    ego.size.w = 1.5f;
    ego.size.l = 3.0f;
    ego.size.h = 2.0f;
    ego.z_rotation = 0;

    auto points = transformation_predictions(transform_matrix_.data(), {ego});
    size_t num = points.size();
    for (size_t i = 0; i < num; ++i) {
      auto& item = points[i];
      auto& corners = std::get<0>(item);
      auto label = std::get<1>(item);
      auto score = std::get<2>(item);
      const int idx_of_line[][2] = {
          {0, 1}, {1, 2}, {2, 3}, {3, 0}, {4, 5}, {5, 6}, {6, 7}, {7, 4}, {0, 4}, {1, 5}, {2, 6}, {3, 7},
      };

      float size = std::sqrt(std::pow(corners[6].x - corners[0].x, 2) + std::pow(corners[6].y - corners[0].y, 2));
      float minx = param_.image_width;
      float miny = param_.image_height;
      for (size_t ioff = 0; ioff < sizeof(idx_of_line) / sizeof(idx_of_line[0]); ++ioff) {
        auto& p0 = corners[idx_of_line[ioff][0]];
        auto& p1 = corners[idx_of_line[ioff][1]];
        minx = std::min(minx, std::min(p0.x, p1.x));
        miny = std::min(miny, std::min(p0.y, p1.y));
        cuosd_draw_line(cuosd_, p0.x, p0.y, p1.x, p1.y, 5, {0, 255, 0, 255});
      }
    }
  }

  virtual void draw_prediction(const std::vector<Prediction>& predictions, bool take_title) override {
    auto points = transformation_predictions(transform_matrix_.data(), predictions);
    size_t num = points.size();
    for (size_t i = 0; i < num; ++i) {
      auto& item = points[i];
      auto& corners = std::get<0>(item);
      auto label = std::get<1>(item);
      auto score = std::get<2>(item);
      const int idx_of_line[][2] = {
          {0, 1}, {1, 2}, {2, 3}, {3, 0}, {4, 5}, {5, 6}, {6, 7}, {7, 4}, {0, 4}, {1, 5}, {2, 6}, {3, 7},
      };

      NameAndColor* name_color = &default_name_color_;
      if (label >= 0 && label < static_cast<int>(param_.classes.size())) {
        name_color = &param_.classes[label];
      }

      float size = std::sqrt(std::pow(corners[6].x - corners[0].x, 2) + std::pow(corners[6].y - corners[0].y, 2));
      float minx = param_.image_width;
      float miny = param_.image_height;
      for (size_t ioff = 0; ioff < sizeof(idx_of_line) / sizeof(idx_of_line[0]); ++ioff) {
        auto& p0 = corners[idx_of_line[ioff][0]];
        auto& p1 = corners[idx_of_line[ioff][1]];
        minx = std::min(minx, std::min(p0.x, p1.x));
        miny = std::min(miny, std::min(p0.y, p1.y));
        cuosd_draw_line(cuosd_, p0.x, p0.y, p1.x, p1.y, 5, {name_color->r, name_color->g, name_color->b, 255});
      }

      if (take_title) {
        size = std::max(size * 0.02f, 5.0f);
        auto title = nv::format("%s %.2f", name_color->name.c_str(), score);
        cuosd_draw_text(cuosd_, title.c_str(), size, UseFont, minx, miny, {name_color->r, name_color->g, name_color->b, 255},
                        {255, 255, 255, 200});
      }
    }
  }

  virtual void apply(unsigned char* image_rgb_device, void* stream) override {
    for (size_t i = 0; i < draw_point_cmds_.size(); ++i) {
      auto& item = draw_point_cmds_[i];
      cuda_linear_launch(draw_point_to, static_cast<hipStream_t>(stream), item.number_of_points,
                         reinterpret_cast<const half5*>(item.points_device), transform_matrix_device_, image_rgb_device,
                         param_.image_width, param_.image_stride, param_.image_height);
    }
    draw_point_cmds_.clear();

    cuosd_apply(cuosd_, image_rgb_device, nullptr, param_.image_width, param_.image_stride, param_.image_height,
                cuOSDImageFormat::RGB, stream);
  }

 private:
  std::vector<BEVArtistDrawPointCommand> draw_point_cmds_;
  std::vector<nvtype::Float4> transform_matrix_;
  float4* transform_matrix_device_ = nullptr;
  cuOSDContext_t cuosd_ = nullptr;
  BEVArtistParameter param_;
  NameAndColor default_name_color_{"Unknow", 0, 0, 0};
};

std::shared_ptr<BEVArtist> create_bev_artist(const BEVArtistParameter& param) {
  std::shared_ptr<BEVArtistImplement> instance(new BEVArtistImplement());
  if (!instance->init(param)) {
    printf("Failed to create BEVArtist\n");
    instance.reset();
  }
  return instance;
}

static __device__ uchar3 load_pixel(const unsigned char* image, int x, int y, float sx, float sy, int width, int stride,
                                    int height) {
  uchar3 rgb[4];
  float src_x = (x + 0.5f) * sx - 0.5f;
  float src_y = (y + 0.5f) * sy - 0.5f;
  int y_low = floorf(src_y);
  int x_low = floorf(src_x);
  int y_high = limit(y_low + 1, 0, height - 1);
  int x_high = limit(x_low + 1, 0, width - 1);
  y_low = limit(y_low, 0, height - 1);
  x_low = limit(x_low, 0, width - 1);

  int ly = rint((src_y - y_low) * INTER_RESIZE_COEF_SCALE);
  int lx = rint((src_x - x_low) * INTER_RESIZE_COEF_SCALE);
  int hy = INTER_RESIZE_COEF_SCALE - ly;
  int hx = INTER_RESIZE_COEF_SCALE - lx;

  rgb[0] = *(uchar3*)&image[y_low * stride + x_low * 3];
  rgb[1] = *(uchar3*)&image[y_low * stride + x_high * 3];
  rgb[2] = *(uchar3*)&image[y_high * stride + x_low * 3];
  rgb[3] = *(uchar3*)&image[y_high * stride + x_high * 3];

  uchar3 output;
  output.x =
      (((hy * ((hx * rgb[0].x + lx * rgb[1].x) >> 4)) >> 16) + ((ly * ((hx * rgb[2].x + lx * rgb[3].x) >> 4)) >> 16) + 2) >> 2;
  output.y =
      (((hy * ((hx * rgb[0].y + lx * rgb[1].y) >> 4)) >> 16) + ((ly * ((hx * rgb[2].y + lx * rgb[3].y) >> 4)) >> 16) + 2) >> 2;
  output.z =
      (((hy * ((hx * rgb[0].z + lx * rgb[1].z) >> 4)) >> 16) + ((ly * ((hx * rgb[2].z + lx * rgb[3].z) >> 4)) >> 16) + 2) >> 2;
  return output;
}

static __global__ void resize_to_kernel(int nx, int ny, int nz, int x0, int y0, float sx, float sy, const unsigned char* img,
                                        int image_width, int image_stride, int image_height, float alpha, unsigned char* output,
                                        int output_stride) {
  int ox = cuda_2d_x;
  int oy = cuda_2d_y;
  if (ox >= nx || oy >= ny) return;

  uchar3 pixel = load_pixel(img, ox, oy, sx, sy, image_width, image_stride, image_height);
  auto& old = *(uchar3*)(output + output_stride * (oy + y0) + (ox + x0) * 3);
  old = make_uchar3(limit(pixel.x * alpha + old.x * (1.0f - alpha), 0.0f, 255.0f),
                    limit(pixel.y * alpha + old.y * (1.0f - alpha), 0.0f, 255.0f),
                    limit(pixel.z * alpha + old.z * (1.0f - alpha), 0.0f, 255.0f));
}

static __global__ void flipx_kernel(int nx, int ny, int nz, const unsigned char* img, int image_stride, unsigned char* output,
                                    int output_stride) {
  int ox = cuda_2d_x;
  int oy = cuda_2d_y;
  if (ox >= nx || oy >= ny) return;

  *(uchar3*)&output[oy * output_stride + ox * 3] = *(uchar3*)&img[oy * image_stride + (nx - ox - 1) * 3];
}

class SceneArtistImplement : public SceneArtist {
 public:
  virtual ~SceneArtistImplement() {
    if (cuosd_) cuosd_context_destroy(cuosd_);
  }

  bool init(const SceneArtistParameter& param) {
    this->param_ = param;
    cuosd_ = cuosd_context_create();
    return cuosd_ != nullptr;
  }

  virtual void flipx(const unsigned char* image_device, int image_width, int image_stride, int image_height,
                     unsigned char* output_device, int output_stride, void* stream) override {
    hipStream_t _stream = static_cast<hipStream_t>(stream);
    cuda_2d_launch(flipx_kernel, _stream, image_width, image_height, 1, image_device, image_stride, output_device, output_stride);
  }

  virtual void resize_to(const unsigned char* image, int x0, int y0, int x1, int y1, int image_width, int image_stride,
                         int image_height, float alpha, void* stream) override {
    x0 = limit(x0, 0, param_.width - 1);
    y0 = limit(y0, 0, param_.height - 1);
    x1 = limit(x1, 1, param_.width);
    y1 = limit(y1, 1, param_.height);
    int ow = x1 - x0;
    int oh = y1 - y0;
    if (ow <= 0 || oh <= 0) return;

    float sx = image_width / (float)ow;
    float sy = image_height / (float)oh;
    hipStream_t _stream = static_cast<hipStream_t>(stream);
    cuda_2d_launch(resize_to_kernel, _stream, ow, oh, 1, x0, y0, sx, sy, image, image_width, image_stride, image_height, alpha,
                   param_.image_device, param_.stride);
  }

 private:
  SceneArtistParameter param_;
  cuOSDContext_t cuosd_ = nullptr;
};

std::shared_ptr<SceneArtist> create_scene_artist(const SceneArtistParameter& param) {
  std::shared_ptr<SceneArtistImplement> instance(new SceneArtistImplement());
  if (!instance->init(param)) {
    printf("Failed to create SceneArtist\n");
    instance.reset();
  }
  return instance;
}

};  // namespace nv