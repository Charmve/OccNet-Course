#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_fp16.h>

#include "common/check.hpp"
#include "common/launch.cuh"
#include "lidar-voxelization.hpp"

namespace bevfusion {
namespace lidar {

static __device__ inline uint64_t hash(uint64_t k) {
  k ^= k >> 16;
  k *= 0x85ebca6b;
  k ^= k >> 13;
  k *= 0xc2b2ae35;
  k ^= k >> 16;
  return k;
}

static __device__ inline void insert_to_hash_table(const uint32_t key, uint32_t *value, const uint32_t hash_size,
                                                   uint32_t *hash_table) {
  uint64_t hash_value = hash(key);
  uint32_t slot = hash_value % (hash_size / 2) /*key, value*/;
  uint32_t empty_key = UINT32_MAX;
  while (true) {
    uint32_t pre_key = atomicCAS(hash_table + slot, empty_key, key);
    if (pre_key == empty_key) {
      hash_table[slot + hash_size / 2 /*offset*/] = atomicAdd(value, 1);
      break;
    } else if (pre_key == key) {
      break;
    }
    slot = (slot + 1) % (hash_size / 2);
  }
}

static __device__ inline uint32_t lookup_hash_table(const uint32_t key, const uint32_t hash_size, const uint32_t *hash_table) {
  uint64_t hash_value = hash(key);
  uint32_t slot = hash_value % (hash_size / 2) /*key, value*/;
  uint32_t empty_key = UINT32_MAX;
  int cnt = 0;
  while (true /* need to be adjusted according to data*/) {
    cnt++;
    if (hash_table[slot] == key) {
      return hash_table[slot + hash_size / 2];
    } else if (hash_table[slot] == empty_key) {
      return empty_key;
    } else {
      slot = (slot + 1) % (hash_size / 2);
    }
  }
  return empty_key;
}

static __global__ void build_hash_table_kernel(size_t points_size, const half *points, VoxelizationParameter param,
                                               unsigned int *hash_table, unsigned int *real_voxel_num) {
  int point_idx = cuda_linear_index;
  if (point_idx >= points_size) return;

  float px = points[param.num_feature * point_idx];
  float py = points[param.num_feature * point_idx + 1];
  float pz = points[param.num_feature * point_idx + 2];

  int voxel_idx = floorf((px - param.min_range.x) / param.voxel_size.x);
  if (voxel_idx < 0 || voxel_idx >= param.grid_size.x) return;

  int voxel_idy = floorf((py - param.min_range.y) / param.voxel_size.y);
  if (voxel_idy < 0 || voxel_idy >= param.grid_size.y) return;

  int voxel_idz = floorf((pz - param.min_range.z) / param.voxel_size.z);
  if (voxel_idz < 0 || voxel_idz >= param.grid_size.z) return;
  unsigned int voxel_offset = (voxel_idz * param.grid_size.y + voxel_idy) * param.grid_size.x + voxel_idx;
  insert_to_hash_table(voxel_offset, real_voxel_num, points_size * 2 * 2, hash_table);
}

template <CoordinateOrder order>
static __device__ void save_result_by_order(uint4 *output, uint x, uint y, uint z);

template <>
__device__ void save_result_by_order<CoordinateOrder::XYZ>(uint4 *output, uint x, uint y, uint z) {
  *output = make_uint4(0, x, y, z);
}

template <>
__device__ void save_result_by_order<CoordinateOrder::ZYX>(uint4 *output, uint x, uint y, uint z) {
  *output = make_uint4(0, z, y, x);
}

template <CoordinateOrder order>
static __global__ void voxelization_kernel(size_t points_size, const half *points, VoxelizationParameter param,
                                           unsigned int *hash_table, unsigned int *num_points_per_voxel, float *voxels_temp,
                                           unsigned int *voxel_indices) {
  int point_idx = cuda_linear_index;
  if (point_idx >= points_size) return;

  float px = points[param.num_feature * point_idx];
  float py = points[param.num_feature * point_idx + 1];
  float pz = points[param.num_feature * point_idx + 2];

  if (px < param.min_range.x || px >= param.max_range.x || py < param.min_range.y || py >= param.max_range.y ||
      pz < param.min_range.z || pz >= param.max_range.z) {
    return;
  }

  int voxel_idx = floorf((px - param.min_range.x) / param.voxel_size.x);
  int voxel_idy = floorf((py - param.min_range.y) / param.voxel_size.y);
  int voxel_idz = floorf((pz - param.min_range.z) / param.voxel_size.z);
  if ((voxel_idx < 0 || voxel_idx >= param.grid_size.x)) {
    return;
  }
  if ((voxel_idy < 0 || voxel_idy >= param.grid_size.y)) {
    return;
  }
  if ((voxel_idz < 0 || voxel_idz >= param.grid_size.z)) {
    return;
  }

  unsigned int voxel_offset = (voxel_idz * param.grid_size.y + voxel_idy) * param.grid_size.x + voxel_idx;

  // scatter to voxels
  unsigned int voxel_id = lookup_hash_table(voxel_offset, points_size * 2 * 2, hash_table);
  if (voxel_id >= param.max_voxels) {
    return;
  }

  unsigned int current_num = atomicAdd(num_points_per_voxel + voxel_id, 1);
  if (current_num < param.max_points_per_voxel) {
    unsigned int dst_offset = voxel_id * (param.num_feature * param.max_points_per_voxel) + current_num * param.num_feature;
    unsigned int src_offset = point_idx * param.num_feature;
    for (int feature_idx = 0; feature_idx < param.num_feature; ++feature_idx) {
      voxels_temp[dst_offset + feature_idx] = points[src_offset + feature_idx];
    }

    // now only deal with batch_size = 1
    // since not sure what the input format will be if batch size > 1
    save_result_by_order<order>(&((uint4 *)voxel_indices)[voxel_id], voxel_idx, voxel_idy, voxel_idz);
  }
}

static __global__ void reduce_mean_kernel(size_t num_voxels, float *voxels_temp, unsigned int *num_points_per_voxel,
                                          int max_points_per_voxel, int feature_num, half *voxel_features) {
  int voxel_idx = cuda_linear_index;
  if (voxel_idx >= num_voxels) return;

  num_points_per_voxel[voxel_idx] =
      num_points_per_voxel[voxel_idx] > max_points_per_voxel ? max_points_per_voxel : num_points_per_voxel[voxel_idx];
  int valid_points_num = num_points_per_voxel[voxel_idx];
  int offset = voxel_idx * max_points_per_voxel * feature_num;
  for (int feature_idx = 0; feature_idx < feature_num; ++feature_idx) {
    for (int point_idx = 0; point_idx < valid_points_num - 1; ++point_idx) {
      voxels_temp[offset + feature_idx] += voxels_temp[offset + (point_idx + 1) * feature_num + feature_idx];
    }
    voxels_temp[offset + feature_idx] /= valid_points_num;
  }

  // move to be continuous
  for (int feature_idx = 0; feature_idx < feature_num; ++feature_idx) {
    int dst_offset = voxel_idx * feature_num;
    int src_offset = voxel_idx * feature_num * max_points_per_voxel;
    voxel_features[dst_offset + feature_idx] = __float2half(voxels_temp[src_offset + feature_idx]);
  }
}

nvtype::Int3 VoxelizationParameter::compute_grid_size(const nvtype::Float3 &max_range, const nvtype::Float3 &min_range,
                                                      const nvtype::Float3 &voxel_size) {
  nvtype::Int3 size;
  size.x = static_cast<int>(std::round((max_range.x - min_range.x) / voxel_size.x));
  size.y = static_cast<int>(std::round((max_range.y - min_range.y) / voxel_size.y));
  size.z = static_cast<int>(std::round((max_range.z - min_range.z) / voxel_size.z));
  return size;
}

class VoxelizationImplement : public Voxelization {
 public:
  virtual ~VoxelizationImplement() {
    if (hash_table_) checkRuntime(hipFree(hash_table_));
    if (voxels_temp_) checkRuntime(hipFree(voxels_temp_));

    if (d_voxel_features_) checkRuntime(hipFree(d_voxel_features_));
    if (d_voxel_num_) checkRuntime(hipFree(d_voxel_num_));
    if (d_voxel_indices_) checkRuntime(hipFree(d_voxel_indices_));

    if (d_real_num_voxels_) checkRuntime(hipFree(d_real_num_voxels_));
    if (h_real_num_voxels_) checkRuntime(hipHostFree(h_real_num_voxels_));
  }

  bool init(VoxelizationParameter param) {
    this->param_ = param;
    this->output_grid_size_ = {(int)param_.grid_size.x, (int)param_.grid_size.y, (int)param_.grid_size.z + 1};

    this->hash_table_size_ = param_.max_points * 2 * 2 * sizeof(unsigned int);
    this->voxels_temp_size_ = param_.max_voxels * param_.max_points_per_voxel * param_.num_feature * sizeof(float);
    this->voxel_features_size_ = param_.max_voxels * param_.max_points_per_voxel * param_.num_feature * sizeof(half);
    this->voxel_num_size_ = param_.max_voxels * sizeof(unsigned int);
    this->voxel_idxs_size_ = param_.max_voxels * 4 * sizeof(unsigned int);

    checkRuntime(hipMalloc(&hash_table_, hash_table_size_));
    checkRuntime(hipMalloc(&voxels_temp_, voxels_temp_size_));
    checkRuntime(hipMalloc(&d_voxel_features_, voxel_features_size_));
    checkRuntime(hipMalloc(&d_voxel_num_, voxel_num_size_));
    checkRuntime(hipMalloc(&d_voxel_indices_, voxel_idxs_size_));
    checkRuntime(hipMalloc(&d_real_num_voxels_, sizeof(unsigned int)));
    checkRuntime(hipHostMalloc(&h_real_num_voxels_, sizeof(unsigned int)));
    return true;
  }

  // points and voxels must be of half type
  virtual void forward(const nvtype::half *points, int num_points, void *stream, CoordinateOrder output_order) override {
    hipStream_t _stream = reinterpret_cast<hipStream_t>(stream);
    const half *_points = reinterpret_cast<const half *>(points);
    checkRuntime(hipMemsetAsync(hash_table_, 0xff, hash_table_size_, _stream));
    checkRuntime(hipMemsetAsync(voxels_temp_, 0xff, voxels_temp_size_, _stream));
    checkRuntime(hipMemsetAsync(d_voxel_num_, 0, voxel_num_size_, _stream));
    checkRuntime(hipMemsetAsync(d_real_num_voxels_, 0, sizeof(unsigned int), _stream));
    cuda_linear_launch(build_hash_table_kernel, _stream, num_points, _points, param_, hash_table_, d_real_num_voxels_);
    checkRuntime(hipMemcpyAsync(h_real_num_voxels_, d_real_num_voxels_, sizeof(int), hipMemcpyDeviceToHost, _stream));

    // for difference output order
    if (output_order == CoordinateOrder::XYZ) {
      cuda_linear_launch(voxelization_kernel<CoordinateOrder::XYZ>, _stream, num_points, _points, param_, hash_table_,
                         d_voxel_num_, voxels_temp_, d_voxel_indices_);
      this->output_grid_size_ = {(int)param_.grid_size.x, (int)param_.grid_size.y, (int)param_.grid_size.z + 1};
    } else if (output_order == CoordinateOrder::ZYX) {
      cuda_linear_launch(voxelization_kernel<CoordinateOrder::ZYX>, _stream, num_points, _points, param_, hash_table_,
                         d_voxel_num_, voxels_temp_, d_voxel_indices_);
      this->output_grid_size_ = {(int)param_.grid_size.z + 1, (int)param_.grid_size.y, (int)param_.grid_size.x};
    } else
      Assertf(false, "Invalid output_order: %d", static_cast<int>(output_order));

    checkRuntime(hipStreamSynchronize(_stream));

    real_num_voxels_ = *h_real_num_voxels_;
    cuda_linear_launch(reduce_mean_kernel, _stream, real_num_voxels_, voxels_temp_, d_voxel_num_, param_.max_points_per_voxel,
                       param_.num_feature, d_voxel_features_);
  }

  virtual unsigned int num_voxels() override { return real_num_voxels_; }

  virtual unsigned int voxel_dim() override { return param_.num_feature; }

  virtual unsigned int indices_dim() override { return 4; }

  virtual std::vector<int> grid_size() override { return output_grid_size_; }

  virtual const void *indices() override { return d_voxel_indices_; }

  virtual const void *features() override { return d_voxel_features_; }

  virtual CoordinateOrder order() override { return order_; }

 private:
  CoordinateOrder order_ = CoordinateOrder::NoneOrder;
  VoxelizationParameter param_;
  unsigned int real_num_voxels_ = 0;
  std::vector<int> output_grid_size_;

  unsigned int *hash_table_ = nullptr;
  float *voxels_temp_ = nullptr;
  unsigned int *d_real_num_voxels_ = nullptr;
  unsigned int *h_real_num_voxels_ = nullptr;
  unsigned int *d_voxel_num_ = nullptr;
  half *d_voxel_features_ = nullptr;
  unsigned int *d_voxel_indices_ = nullptr;
  unsigned int hash_table_size_;
  unsigned int voxels_temp_size_;
  unsigned int voxel_features_size_;
  unsigned int voxel_idxs_size_;
  unsigned int voxel_num_size_;
};

std::shared_ptr<Voxelization> create_voxelization(VoxelizationParameter param) {
  std::shared_ptr<VoxelizationImplement> impl(new VoxelizationImplement());
  if (!impl->init(param)) {
    impl.reset();
  }
  return impl;
}

};  // namespace lidar
};  // namespace bevfusion
